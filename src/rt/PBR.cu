#include "hip/hip_runtime.h"
#include "PBR.h"

namespace cg::rt
{

__host__ __device__
inline vec3 BRDF_diffuse(const Material& m)
{
    // Lambertian diffuse
    return m.diffuse * std::numbers::inv_pi_v<float>;
}

__host__ __device__
inline vec3 BRDF_specular(
    const vec3& L,
    const vec3& V,
    const vec3& N,
    float dotNV,
    float dotNL,
    float roughness,
    const vec3& R0)
{
    vec3 H = (L + V).versor();
    return schlick(R0, vec3::dot(L, H))
        * BRDF_microfacet(dotNV, dotNL, vec3::dot(H, N), roughness);
}

__host__ __device__
vec3 BRDF(
    const vec3& I,
    const vec3& L,
    const vec3& V,
    const vec3& N,
    float dotNV,
    float dotNL,
    const Material& m)
{
    vec3 d = BRDF_diffuse(m);
    vec3 s = BRDF_specular(L, V, N, dotNV, dotNL, m.roughness, m.specular);
    return I * mix(d, s, m.metalness) * dotNL;
}

__host__ __device__
bool lightVector(float& d, vec3& L, const vec3& P, const Light& light)
{
    if (light.isDirectional())
    {
        L = -light.direction;
        d = numeric_limits<float>::infinity();
        return true;
    }
    L = light.position - P;
    d = L.length();

    if (d < 1e-14f || (light.range > 0 && d > light.range))
        return false;

    L *= (1 / d);
    if (light.isPoint())
        return true;

    // spot
    float DL = vec3::dot(light.direction, L);
    return DL < 0 && light.angle > 2 * acosf(DL);
}

__host__ __device__
vec3 lightColor(float d, const Light& light)
{
    // directional light
    if (light.isDirectional())
        return light.color;

    float range = light.range;
    float f;
    if (range == 0) // infinite range
    {
        f = 1 / d;
        f *= f;
    }
    else
    {
        f = d / range;
        f = 1 + f * (f - 2); // (1 - f)^2
    }
    return light.color * f;
}

} // namespace cg::rt
